#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include "opencv2/core/core.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/video/background_segm.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/legacy/legacy.hpp"
#include <opencv2/gpu/gpumat.hpp> 
#include <opencv2/gpu/gpu.hpp>
#include "math.h"

using namespace cv;
using namespace std;

__global__ void overlayImageKernel(const gpu::PtrStepSz<uchar> background, const gpu::PtrStepSz<uchar> foreground, gpu::PtrStepSz<uchar> output)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	double opacity = ((double)foreground.data[y * foreground.step + x * 4 + 3]) / 255.;

	for (int c = 0; opacity > 0 && c < 3; ++c)
	{
		unsigned char foregroundPx = foreground.data[y * foreground.step + x * 4 + c];
		unsigned char backgroundPx = background.data[y * background.step + x * 3 + c];
		(output).data[y* (output).step + 3*x + c] = (uchar)(backgroundPx * (1. - opacity) + foregroundPx * opacity);
	}
}

__global__ void overlayImage2Kernel(const unsigned char *foreground, unsigned char *output)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	double opacity = ((double)foreground[y * 2560 + x * 4 + 3]) / 255.;

	for (int c = 0; opacity > 0 && c < 3; ++c)
	{
		unsigned char foregroundPx = foreground[y * 2560 + x * 4 + c];
		unsigned char backgroundPx = output[y * 1920 + x * 3 + c];
		output[y * 1920 + 3 * x + c] = (uchar)(backgroundPx * (1. - opacity) + foregroundPx * opacity);
	}
}

__global__ void similarity_Kernel(double *a, double *b, double *outputArray)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int i = tid + bid * blockDim.x;
	__shared__ double s_data[1024], colorArray[4096];

	colorArray[i] = sqrt(a[i] * b[i]);

	s_data[tid] = colorArray[bid * 1024 + tid];
	__syncthreads();

	for (int i = 512; i > 0; i /= 2)
	{
		if (tid < i)
			s_data[tid] = s_data[tid] + s_data[tid + i];
		__syncthreads();
	}
	if (tid == 0)
	{
		outputArray[bid] = s_data[0];
	}
}

__global__ void computeHistKernel(const gpu::PtrStepSz<uchar> roiMat, double *kernel, double *histOutput2)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	__shared__ double histOutput[4096];
	//double kernel_sum;

	if (kernel[x + y * roiMat.cols] == 0){}
	else
	{
		int val0 = roiMat.data[y* roiMat.step + 3 * x];
		int val1 = roiMat.data[y* roiMat.step + 3 * x + 1];
		int val2 = roiMat.data[y* roiMat.step + 3 * x + 2];
		int idx = (val0 / 16) * 256 + (val1 / 16) * 16 + val2 / 16;
		histOutput[idx] = histOutput[idx] + kernel[x + y * roiMat.cols];
		//kernel_sum = kernel_sum + kernel[x + y * roiMat.cols];
	}
}

__global__ void MatrixMulKernel(int *a, int *b, int *c)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int tid = x + y*blockDim.x * gridDim.x;
	int width = blockDim.x * gridDim.x;
	int Pvalue = 0;
	for (int k = 0; k < width; k++)
		Pvalue = Pvalue + a[y * width + k] * b[k * width + x];

	c[tid] = Pvalue;
}

int parallel_similarity(double* hist, double* hist2, double &similarity)
{
	similarity = 0.0;

	const int arraySize = 4096;
	double d[4];
	double *dev_a = 0;
	double *dev_b = 0;
	double *dev_d = 0;

	hipMalloc((void**)&dev_a, arraySize * sizeof(double));
	hipMalloc((void**)&dev_b, arraySize * sizeof(double));
	hipMalloc((void**)&dev_d, 4 * sizeof(double));

	hipMemcpy(dev_a, hist, arraySize * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, hist2, arraySize * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_d, d, 4 * sizeof(double), hipMemcpyHostToDevice);

	double t = 0.0;
	t = (double)cvGetTickCount();

	similarity_Kernel << <4, 1024 >> >(dev_a, dev_b, dev_d);

	t = (double)cvGetTickCount() - t;
	printf("similarity_Kernel = %gms\n", t / ((double)cvGetTickFrequency() *1000.));

	hipMemcpy(d, dev_d, 4 * sizeof(double), hipMemcpyDeviceToHost);
	for (int i = 0; i < 4; i++)
	{
		similarity += d[i];
	}
	return 0;
}

void parallel_overlayImage(const cv::Mat &background, const cv::Mat &foreground, cv::Mat &output, int setting)
{
	background.copyTo(output);

	if (setting == 1) // Use "upload & download" API
	{
		gpu::GpuMat gpu_background, gpu_foreground, gpu_output;
		gpu_background.upload(background);
		gpu_output.upload(output);
		gpu_foreground.upload(foreground);

		dim3 blocks(1, background.rows);
		dim3 threads(background.cols, 1);
		overlayImageKernel << < blocks, threads >> > (gpu_background, gpu_foreground, gpu_output);

		gpu_output.download(output);
	}
	if (setting == 2) // Use "hipMemcpy"
	{
		unsigned char *dev_foreground = 0;
		unsigned char *dev_output = 0;
		const int b_Size = background.cols * background.rows * 3;
		const int f_Size = background.cols * background.rows * 4;

		hipMalloc((void**)&dev_foreground, f_Size * sizeof(unsigned char));
		hipMalloc((void**)&dev_output, b_Size * sizeof(unsigned char));
		hipMemcpy(dev_foreground, foreground.data, f_Size * sizeof(unsigned char), hipMemcpyHostToDevice);
		hipMemcpy(dev_output, output.data, b_Size * sizeof(unsigned char), hipMemcpyHostToDevice);

		dim3 blocks(1, background.rows);
		dim3 threads(background.cols, 1);
		overlayImage2Kernel << < blocks, threads >> > (dev_foreground, dev_output);

		hipMemcpy(output.data, dev_output, b_Size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	}
}

void parallel_computeHist(const Mat &roiMat, double *hist)
{
	double kernel[20000];
	int H = roiMat.rows - 1;
	int W = roiMat.cols - 1;
	int w = W / 2, h = H / 2;
	int kernelSize = roiMat.rows * roiMat.cols;
	double histTest[4096];
	memset(hist, 0, 4096 * sizeof(double)); // reset hist to 0
	memset(kernel, 0, 20000 * sizeof(double)); // reset hist to 0
	double kernel_sum = 0; // sum for normalize

	if (roiMat.data == NULL) return;
	int n = 0;
	for (int y = 0; y < roiMat.rows; y++)
	{
		for (int x = 0; x < roiMat.cols; x++)
		{
			// scale to unit circle
			float dist_y = (float)(y - h) / h;
			float dist_x = (float)(x - w) / w;
			float distToCen = dist_x*dist_x + dist_y*dist_y; // distance from (i, j) to bbs center

			if (distToCen >= 1)
				kernel[x + y * roiMat.cols] = 0.0;
			else
				kernel[x + y * roiMat.cols] = 2 * (1 - distToCen) * 0.31831;

			if (kernel[x + y * roiMat.cols] == 0){}
			else
			{
				int val0 = roiMat.data[y* roiMat.step + 3 * x];
				int val1 = roiMat.data[y* roiMat.step + 3 * x + 1];
				int val2 = roiMat.data[y* roiMat.step + 3 * x + 2];
				int idx = (val0 / 16) * 256 + (val1 / 16) * 16 + val2 / 16;
				hist[idx] += kernel[x + y * roiMat.cols];
				kernel_sum += kernel[x + y * roiMat.cols];
			}
		}
	}

/*	gpu::GpuMat gpu_roiMat;
	gpu_roiMat.upload(roiMat);

	double *dev_h = 0;
	hipMalloc((void**)&dev_h, 4096 * sizeof(double));
	hipMemcpy(dev_h, hist, 4096 * sizeof(double), hipMemcpyHostToDevice);

	double *dev_k = 0;
	hipMalloc((void**)&dev_k, kernelSize * sizeof(double));
	hipMemcpy(dev_k, kernel, kernelSize * sizeof(double), hipMemcpyHostToDevice);

	dim3 blocks(1, roiMat.rows);
	dim3 threads(roiMat.cols, 1);
	computeHistKernel << < blocks, threads >> > (gpu_roiMat, dev_k, dev_h);

	hipMemcpy(hist, dev_h, 4096 * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(kernel, dev_k, 4096 * sizeof(double), hipMemcpyDeviceToHost);*/

	for (int i = 0; i < 4096;i++)
	{
		histTest[i] = hist[i];
	}
	for (int i = 0; i < 4096; i++)
	{
		hist[i] /= kernel_sum;
	}
}

